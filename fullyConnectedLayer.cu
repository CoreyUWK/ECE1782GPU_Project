
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <math.h>

#define INPUT_SIZE 22400
#define OUTPUT_SIZE 256

/*You can use the following for any CUDA function that returns cudaError_t type*/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code == hipSuccess) return;

    fprintf(stderr,"Error: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
}

/*Use the following to get a timestamp*/
double getTimeStamp() {
    struct timeval tv;
    gettimeofday( &tv, NULL );
    return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

void initArray(float *A, int n) {
    for (int i = 0; i < n; i++) {
        A[i] = rand() % 10;
    }
}

void initWeights(float *W, int inSize, int outSize) {
    for (int i = 0; i < inSize; i++) {
        for (int j = 0; j < outSize; j++) {
            int offset = (i * outSize) + j;
            W[offset] = rand() % 10;
        }
    }
}

void printArray(float *A, int n) {
    for (int i = 0; i < n; i++) {
        printf("%f  ", A[i]);
    }
    printf("\n");
}

void printWeights(float *W, int n, int m) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            int offset = i * m + j;
            printf("%f  ", W[offset]);
        }
        printf("\n");
    }
}

void host_FC(float *output, float *input, float *W, float *b, int inSize, int outSize) {
    for (int j = 0; j < outSize; j++) {
        for (int i = 0; i < inSize; i++) {
            int offset = i * outSize + j;
            output[j] += input[i] * W[offset];
        }
    }
    for (int j = 0; j < outSize; j++) {
        output[j] += b[j];
    }
}

// __constant__ float device_input[INPUT_SIZE];
// __constant__ float device_b[OUTPUT_SIZE];
__global__ void FC(float *output, float *input, float *W, float *b, int inSize, int outSize) {
// __global__ void FC(float *output, float *W, int inSize, int outSize) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if ((j >= outSize)) {
        return;
    }

    float sum = 0.0;
    for (int i = 0; i < inSize; i++) {
        int offset = i * outSize + j;
        sum += input[i] * W[offset];
        // sum += device_input[i] * W[offset];
    }

    output[j] = sum + b[j];
    // output[j] = sum + device_b[j];
}

int main() {
    // alloc memory host-side
    float *h_in = (float *) malloc (INPUT_SIZE * sizeof(float));
    float *h_W = (float *) malloc (INPUT_SIZE * OUTPUT_SIZE * sizeof(float));
    float *h_b = (float *) malloc (OUTPUT_SIZE * sizeof(float));
    float *h_out = (float *) malloc (OUTPUT_SIZE * sizeof(float)); // host result
    float *h_dout = (float *) malloc (OUTPUT_SIZE * sizeof(float)); // host result

    hipHostRegister(h_in, INPUT_SIZE * sizeof(float), 0);
    hipHostRegister(h_W, INPUT_SIZE * OUTPUT_SIZE * sizeof(float), 0);
    hipHostRegister(h_b, OUTPUT_SIZE * sizeof(float), 0);

    initArray(h_in, INPUT_SIZE);
    initWeights(h_W, INPUT_SIZE, OUTPUT_SIZE);
    initArray(h_b, OUTPUT_SIZE);

    gpuErrchk(hipDeviceReset());
    
    // alloc memory device side
    float *d_in;
    float *d_W;
    float *d_b;
    float *d_out;
    gpuErrchk( hipMalloc( (void **) &d_in, INPUT_SIZE * sizeof(float) ) );
    gpuErrchk( hipMalloc( (void **) &d_W, INPUT_SIZE * OUTPUT_SIZE * sizeof(float) ) );
    gpuErrchk( hipMalloc( (void **) &d_b, OUTPUT_SIZE * sizeof(float) ) );
    gpuErrchk( hipMalloc( (void **) &d_out, OUTPUT_SIZE * sizeof(float) ) );

    // transfer data to device
    gpuErrchk( hipMemcpy(d_in, h_in, INPUT_SIZE * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_W, h_W, INPUT_SIZE * OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_b, h_b, OUTPUT_SIZE * sizeof(float), hipMemcpyHostToDevice) );
    // gpuErrchk( cudaMemcpyToSymbol(device_input, h_in, INPUT_SIZE * sizeof(float)) );
    // gpuErrchk( cudaMemcpyToSymbol(device_b, h_b, OUTPUT_SIZE * sizeof(float)) );

    double start_time = getTimeStamp();

	// invoke kernel
    dim3 block(32, 32); // configure
    dim3 grid((OUTPUT_SIZE+block.x-1)/block.x, (INPUT_SIZE+block.y-1)/block.y);
    FC<<<64, 1024>>>(d_out, d_in, d_W, d_b, INPUT_SIZE, OUTPUT_SIZE);
    // FC<<<grid, block>>>(d_out, d_W, INPUT_SIZE, OUTPUT_SIZE);
    gpuErrchk( hipDeviceSynchronize() );

    // copy data back
    gpuErrchk( hipMemcpy(h_dout, d_out, OUTPUT_SIZE * sizeof(float), hipMemcpyDeviceToHost) );

    double end_time = getTimeStamp();
    int total_time_ms = (int) ceil ((end_time-start_time)*1000);

    host_FC(h_out, h_in, h_W, h_b, INPUT_SIZE, OUTPUT_SIZE);
    // printArray(h_in, INPUT_SIZE);
    // printf("\n");
    // printWeights(h_W, INPUT_SIZE, OUTPUT_SIZE);
    // printf("\n");
    // printArray(h_b, OUTPUT_SIZE);
    // printf("\n");
    // printArray(h_out, OUTPUT_SIZE);
    // printf("\n");
    // printArray(h_dout, OUTPUT_SIZE);
    // printf("\n");

    for (int i = 0; i < OUTPUT_SIZE; i++) {
        if (h_out[i] != h_dout[i]) {
            printf("Error: CPU result and GPU result mismatch at offset: %d.\n", i);
            return 0;
        }
    }

    printf("%d\n", total_time_ms);

    hipHostUnregister(h_in);
    hipHostUnregister(h_W);
    hipHostUnregister(h_b);

    // free gpu resources
    gpuErrchk( hipFree(d_in) );
    gpuErrchk( hipFree(d_W) );
    gpuErrchk( hipFree(d_b) );
    gpuErrchk( hipFree(d_out) );
    gpuErrchk( hipDeviceReset() );

    return 0;

}

